
#include <hip/hip_runtime.h>
extern "C" __global__ void outerProduct
(double *a,
double *b,
double **c,
int N)
{
int row = blockIdx.y*blockDim.y + threadIdx.y; // compute row 
int col = blockIdx.x*blockDim.x + threadIdx.x; // compute column 
if (row < N && col < N)
c[row][col] = a[row]*b[col];
}

